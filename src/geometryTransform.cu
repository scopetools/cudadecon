#include "hip/hip_runtime.h"
#include <GPUBuffer.h>


__global__ void deskew_kernel(float *in, int nx, int ny, int nz,
                              float *out, int nxOut, int extraShift,
                              double deskewFactor, float padVal)
{
  unsigned xout = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned yout = blockIdx.y;
  unsigned zout = blockIdx.z;

  if (xout < nxOut) {
    float xin = (xout - nxOut/2.+extraShift) - deskewFactor*(zout-nz/2.) + nx/2.;

    unsigned indout = zout * nxOut * ny + yout * nxOut + xout;
    if (xin >= 0 && xin < nx-1) {

      // 09-03-2013 Very important lesson learned:
      // the (unsigned int) casting has be placed right there because
      // otherwise, the entire express would evaluate as floating point and
      // there're only 24-bit mantissa, so any odd index that's > 16777216 would
      // inaccurately rounded up. int or unsigned does not have the 24-bit limit.
      unsigned indin = zout * nx * ny + yout * nx + (unsigned int) floor(xin);

      float offset = xin - floor(xin);
      out[indout] = (1-offset) * in[indin] + offset * in[indin+1]; // linear interpolation done within each slice (i.e. along x)
    }
    else
      out[indout] = padVal;
  }
}

__host__ void deskew_GPU(GPUBuffer &inBuf, int nx, int ny, int nz,
                         double deskewFactor, GPUBuffer &outBuf,
                         int newNx, int extraShift, float padVal)
{
  dim3 block(128, 1, 1);
  unsigned nxBlocks = (unsigned ) ceil(newNx / (float) block.x);
  dim3 grid(nxBlocks, ny, nz);

  deskew_kernel<<<grid, block>>>((float *) inBuf.getPtr(),
                                 nx, ny, nz, 
                                 (float *) outBuf.getPtr(), newNx,
                                 extraShift, deskewFactor, padVal);
#ifndef NDEBUG
  std::cout<< "deskew_GPU(): " << hipGetErrorString(hipGetLastError()) << std::endl;
#endif
}

__global__ void rotate_kernel(float *in, int nx_in, int ny, int nz_in,
                              float *out, int nx_out, int nz_out,
                              float *rotMat)
{
  unsigned xout = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned yout = blockIdx.y;
  unsigned zout = blockIdx.z;

  if (xout < nx_out) {
    float xout_centered, zout_centered;
    xout_centered = xout - nx_out/2.;
    zout_centered = zout - nz_out/2.;

    unsigned nxy_in = nx_in * ny;
    unsigned nxy_out = nx_out * ny;
    unsigned yind_out = yout * nx_out;
    unsigned yind_in = yout * nx_in;

    float zin = rotMat[0] * zout_centered + rotMat[1] * xout_centered + nz_in/2.;
    float xin = rotMat[2] * zout_centered + rotMat[3] * xout_centered + nx_in/2.;

    unsigned indout = (nz_out-1-zout) * nxy_out + yind_out + xout; // flip z indices

    if (xin >= 0 && xin < nx_in-1 && zin >=0 && zin < nz_in-1) {

      unsigned indin00 = (unsigned) floor(zin) * nxy_in + yind_in + (unsigned) floor(xin);
      unsigned indin01 = indin00 + 1;
      unsigned indin10 = indin00 + nxy_in;
      unsigned indin11 = indin10 + 1;

      float xoffset = xin - floor(xin);
      float zoffset = zin - floor(zin);
      out[indout] = (1-zoffset) * ( (1-xoffset) * in[indin00] + xoffset * in[indin01]) + 
        zoffset * ((1-xoffset) * in[indin10] + xoffset * in[indin11]);
    }
    else
      out[indout] = 0.f;
  }
}

__host__ void rotate_GPU(GPUBuffer &inBuf, int nx, int ny, int nz,
                         GPUBuffer &rotMatrix, GPUBuffer &outBuf,
                         int nx_out, int nz_out)
{
  dim3 block(128, 1, 1);
  unsigned nxBlocks = (unsigned ) ceil(nx_out / (float) block.x);
  dim3 grid(nxBlocks, ny, nz_out);

  rotate_kernel<<<grid, block>>>((float *) inBuf.getPtr(),
                                 nx, ny, nz,
                                 (float *) outBuf.getPtr(),
                                 nx_out, nz_out,
                                 (float *) rotMatrix.getPtr());
#ifndef NDEBUG
  std::cout<< "rotate_GPU(): " << hipGetErrorString(hipGetLastError()) << std::endl;
#endif
}

__global__ void crop_kernel(float *in, int nx, int ny, int nz,
                            int new_nx, int new_ny, int new_nz,
                            float *out)
{
  unsigned xout = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned yout = blockIdx.y;
  unsigned zout = blockIdx.z;

  if (xout < new_nx) { 
    // Assumption: new dimensions are <= old ones
    unsigned xin = xout + nx - new_nx;
    unsigned yin = yout + ny - new_ny;
    unsigned zin = zout + nz - new_nz;
    unsigned indout = zout * new_nx * new_ny + yout * new_nx + xout;
    unsigned indin = zin * nx * ny + yin * nx + xin;
    out[indout] = in[indin];
  }
}


__host__ void cropGPU(GPUBuffer &inBuf, int nx, int ny, int nz,
                      int new_nx, int new_ny, int new_nz,
                      GPUBuffer &outBuf)
{

  dim3 block(128, 1, 1);
  unsigned nxBlocks = (unsigned ) ceil(new_nx / (float) block.x);
  dim3 grid(nxBlocks, new_ny, new_nz);

  crop_kernel<<<grid, block>>>((float *) inBuf.getPtr(),
                               nx, ny, nz,
                               new_nx, new_ny, new_nz,
                               (float *) outBuf.getPtr());

#ifndef NDEBUG
  std::cout<< "cropGPU(): " << hipGetErrorString(hipGetLastError()) << std::endl;
#endif
}

// ******************************************************************//
// Duplicate the first Z half of the "in" stack, in reverse-Z order,
// into the 2nd Z half of it; essentially faking continuous structure
// in Z to reduce Z ringing from FFT
// ******************************************************************//
__global__ void dupRevStack_kernel(float *in, unsigned nx, unsigned nxy, unsigned nz)
{
  unsigned xin = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned yin = blockIdx.y;
  unsigned zin = blockIdx.z;

  if (xin < nx) {
    unsigned zout = (nz<<1) - zin - 1; // + and - take precedence over << and >>!!
    unsigned indout = zout * nxy + yin * nx + xin;
    unsigned indin  =  zin * nxy + yin * nx + xin;
    in[indout] = in[indin];
  }
}

__host__ void duplicateReversedStack_GPU(GPUBuffer &zExpanded, int nx, int ny, int nz)
{
  dim3 block(128, 1, 1);
  unsigned nxBlocks = (unsigned ) ceil(nx / (float) block.x);
  dim3 grid(nxBlocks, ny, nz);

  dupRevStack_kernel<<<grid, block>>>((float *) zExpanded.getPtr(),
                                      nx, nx*ny, nz);
#ifndef NDEBUG
  std::cout<< "duplicateReversedStack_GPU(): " << hipGetErrorString(hipGetLastError()) << std::endl;
#endif
}


texture<float, hipTextureType3D, hipReadModeElementType> texRef;


// Simple transformation kernel
__global__ void transformKernel(float *output,
                                int nx, int ny, int nz,
                                float *mat)
{

  // Calculate texture coordinates
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

  if (x >= nx || y >= ny || z >= nz) {
    return;
  }

  // for normalized coordinates
  //float u = x / (float)nx;
  //float v = y / (float)ny;
  //float w = z / (float)nz;

  float u = x;
  float v = y;
  float w = z;

  float tu = mat[0]*u + mat[1]*v + mat[2] *w +  mat[3] + 0.5f;
  float tv = mat[4]*u + mat[5]*v + mat[6] *w +  mat[7] + 0.5f;
  float tw = mat[8]*u + mat[9]*v + mat[10]*w + mat[11] + 0.5f;

  // Read from texture and write to global memory
  int idx = z * (nx*ny) + y * nx + x;
  output[idx] = tex3D(texRef, tu, tv, tw);
}

// Simple transformation kernel
__global__ void transformKernelRA(float *output,
                                  int nx, int ny, int nz,
                                  float dx, float dy, float dz,
                                  float *mat)
{

  // Calculate texture coordinates
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

  if (x >= nx || y >= ny || z >= nz) {
    return;
  }

  float u = x;
  float v = y;
  float w = z;

  // intrinsic coords to world
  u = 0.5 + (u - 0.5) * dx;
  v = 0.5 + (v - 0.5) * dy;
  w = 0.5 + (w - 0.5) * dz;

  // transform coordinates in world coordinate frame
  float tu = mat[0]*u + mat[1]*v + mat[2] *w +  mat[3];
  float tv = mat[4]*u + mat[5]*v + mat[6] *w +  mat[7];
  float tw = mat[8]*u + mat[9]*v + mat[10]*w + mat[11];

  // world coords to intrinsic
  tu = 0.5 + (tu - 0.5) / dx;
  tv = 0.5 + (tv - 0.5) / dy;
  tw = 0.5 + (tw - 0.5) / dz;

  // Read from texture and write to global memory
  int idx = z * (nx*ny) + y * nx + x;
  output[idx] = tex3D(texRef, tu, tv, tw);
}


// host data
__host__ void affine_GPU(hipArray *cuArray, int nx, int ny, int nz,
                         float * result, GPUBuffer &affMat)
{

  // Allocate CUDA array in device memory
  hipChannelFormatDesc channelDesc =
    hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

  // Set texture reference parameters
  texRef.addressMode[0] = hipAddressModeBorder;
  texRef.addressMode[1] = hipAddressModeBorder;
  texRef.addressMode[2] = hipAddressModeBorder;
  texRef.filterMode = hipFilterModeLinear;
  texRef.normalized = false;

  // Bind the array to the texture reference
  hipBindTextureToArray(texRef, cuArray, channelDesc);

  // Allocate result of transformation in device memory
  float* output;
  hipMalloc(&output, nx * ny * nz * sizeof(float));

  // Invoke kernel dim3
  dim3 dimBlock(16,16,4);
  dim3 dimGrid((nx + dimBlock.x - 1) / dimBlock.x,
               (ny + dimBlock.y - 1) / dimBlock.y,
               (nz + dimBlock.z - 1) / dimBlock.z);

  transformKernel<<<dimGrid, dimBlock>>>(output, nx, ny, nz, (float *) affMat.getPtr());
  CudaCheckError();

  //transfer result back to host
  hipMemcpy(result, output, nz * nx * ny * sizeof(float), hipMemcpyDeviceToHost);

  // Free device memory
  hipFreeArray(cuArray);
  hipFree(output);
}

// host data
__host__ void affine_GPU_RA(hipArray *cuArray, int nx, int ny, int nz,
                         float dx, float dy, float dz,
                         float * result, GPUBuffer &affMat)
{

    // Allocate CUDA array in device memory
    hipChannelFormatDesc channelDesc =
      hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

    // Set texture reference parameters
    texRef.addressMode[0] = hipAddressModeBorder;
    texRef.addressMode[1] = hipAddressModeBorder;
    texRef.addressMode[2] = hipAddressModeBorder;
    texRef.filterMode = hipFilterModeLinear;
    texRef.normalized = false;

    // Bind the array to the texture reference
    hipBindTextureToArray(texRef, cuArray, channelDesc);

    // Allocate result of transformation in device memory
    float* output;
    hipMalloc(&output, nx * ny * nz * sizeof(float));

    // Invoke kernel dim3
    dim3 dimBlock(16,16,4);
    dim3 dimGrid((nx + dimBlock.x - 1) / dimBlock.x,
                 (ny + dimBlock.y - 1) / dimBlock.y,
                 (nz + dimBlock.z - 1) / dimBlock.z);

    transformKernelRA<<<dimGrid, dimBlock>>>(output, nx, ny, nz, dx, dy, dz, (float *) affMat.getPtr());
    CudaCheckError();

    //transfer result back to host
    hipMemcpy(result, output, nz * nx * ny * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFreeArray(cuArray);
    hipFree(output);
}
