#include "hip/hip_runtime.h"
#include <GPUBuffer.h>

#include "camcor_context.h"
#include "cutilSafeCall.h"

__host__ void setupCamCor(CamcorContext* context, float* h_camparam) {
  if (!context) return;  // check for null pointer

  int nx = context->nx;
  int ny = context->ny;

  // Allocate CUDA array in device memory
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  hipArray* d_camparamArray;
  hipExtent extent = make_hipExtent(nx, ny, 3);
  hipMalloc3DArray(&d_camparamArray, &channelDesc, extent);

  // Copy host camparams to device memory
  hipMemcpy3DParms parms = {0};
  parms.srcPtr = make_hipPitchedPtr(h_camparam, nx * sizeof(float), nx, ny);
  parms.dstArray = d_camparamArray;
  parms.extent = extent;
  parms.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&parms);

  // Create texture object
  hipResourceDesc resDesc = {};
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = d_camparamArray;

  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeBorder;
  texDesc.addressMode[1] = hipAddressModeBorder;
  texDesc.addressMode[2] = hipAddressModeBorder;
  texDesc.filterMode = hipFilterModePoint;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  hipCreateTextureObject(&context->camparam_texObj, &resDesc, &texDesc, NULL);
}

__host__ void setupData(CamcorContext* context, unsigned* h_data) {
  int nx = context->nx;
  int ny = context->ny;
  int nz = context->nz;

  // Allocate CUDA array in device memory
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned>();
  hipArray* d_dataArray;
  hipExtent extent = make_hipExtent(nx, ny, nz);
  hipMalloc3DArray(&d_dataArray, &channelDesc, extent);

  // Copy host data to device memory
  hipMemcpy3DParms parms = {0};
  parms.srcPtr = make_hipPitchedPtr(h_data, nx * sizeof(unsigned), nx, ny);
  parms.dstArray = d_dataArray;
  parms.extent = extent;
  parms.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&parms);

  // Define resource descriptor
  hipResourceDesc resDesc = {};
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = d_dataArray;

  // Define texture descriptor
  hipTextureDesc texDesc = {};
  texDesc.addressMode[0] = hipAddressModeWrap;
  texDesc.addressMode[1] = hipAddressModeWrap;
  texDesc.addressMode[2] = hipAddressModeWrap;
  texDesc.filterMode = hipFilterModePoint;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 1;

  // Create texture object
  hipCreateTextureObject(&context->data_texObj, &resDesc, &texDesc, nullptr);
}

__global__ void camcor_kernel(unsigned nx, unsigned ny, unsigned nz,
                              hipTextureObject_t camparam_texObj, hipTextureObject_t data_texObj,
                              unsigned short* output) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

  if (x >= nx || y >= ny || z >= nz) {
    return;
  }

  // normalize coordinates
  float u = (x + 0.5) / (float)nx;
  float v = (y + 0.5) / (float)ny;
  float w = (z + 0.5) / (float)nz;
  float wp = (z - 0.5) / (float)nz;

  float a = tex3D<float>(camparam_texObj, x, y, 0);
  float b = tex3D<float>(camparam_texObj, x, y, 1);
  float offset = tex3D<float>(camparam_texObj, x, y, 2);

  unsigned voxel = tex3D<unsigned int>(data_texObj, u, v, w);
  unsigned previousvoxel = tex3D<unsigned int>(data_texObj, u, v, wp);

  unsigned int i = z * ny * nx + y * nx + x;
  output[i] =
      (unsigned short)voxel - offset - 0.9f * a * (1 - expf(-b * (previousvoxel - offset)));
  output[i] = output[i] > 0 ? output[i] : 0;
}

__host__ void camcor_GPU(CamcorContext* context, GPUBuffer& outBuf) {
  unsigned nx = context->nx;
  unsigned ny = context->ny;
  unsigned nz = context->nz;

  dim3 dimBlock(16, 16, 1);
  dim3 dimGrid((nx + dimBlock.x - 1) / dimBlock.x, (ny + dimBlock.y - 1) / dimBlock.y,
               (nz + dimBlock.z - 1) / dimBlock.z);

  camcor_kernel<<<dimGrid, dimBlock>>>(nx, ny, nz, context->camparam_texObj, context->data_texObj,
                                       (unsigned short*)outBuf.getPtr());
  // CudaCheckError();
}
