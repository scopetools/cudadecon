#include "hip/hip_runtime.h"
#include <GPUBuffer.h>
#include "cutilSafeCall.h"


__constant__ unsigned const_nx;
__constant__ unsigned const_ny;
__constant__ unsigned const_nz;
__constant__ unsigned const_nxy;
__constant__ unsigned const_nxyz;


texture<float, hipTextureType3D, hipReadModeElementType> camparam_tex;
texture<unsigned, hipTextureType3D, hipReadModeElementType> data_tex;

__host__ void setupConst(int nx, int ny, int nz)
{
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_nx), &nx, sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_ny), &ny, sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_nz), &nz, sizeof(int)));
  unsigned int nxy = nx*ny;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_nxy), &nxy, sizeof(unsigned int)));
  unsigned int nxyz = nx*ny*nz;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_nxyz), &nxyz, sizeof(unsigned int)));
}

__host__ void setupCamCor(int nx, int ny, float * h_caparam)
{

  // Allocate CUDA array in device memory
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc(
      32, 0, 0, 0, hipChannelFormatKindFloat);

  hipArray* d_camparamArray;
  hipExtent extent = make_hipExtent(nx, ny, 3);
  hipMalloc3DArray(
      &d_camparamArray,
      &channelDesc,
      extent,
      hipArrayDefault
  );

  // Copy host camparams to device memory
  hipMemcpy3DParms parms = {0};
  parms.srcPtr = make_hipPitchedPtr(
      h_caparam,
      nx * sizeof(float), nx, ny
  );
  parms.dstArray = d_camparamArray;
  parms.extent = extent;
  parms.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&parms);

  // Set texture reference parameters
  camparam_tex.addressMode[0] = hipAddressModeBorder;
  camparam_tex.addressMode[1] = hipAddressModeBorder;
  camparam_tex.addressMode[2] = hipAddressModeBorder;
  camparam_tex.filterMode = hipFilterModePoint;
  camparam_tex.normalized = false;

  // Bind the array to the texture reference
  hipBindTextureToArray(camparam_tex, d_camparamArray, channelDesc);
}

__host__ void setupData(int nx, int ny, int nz, unsigned * h_data)
{

  hipArray* d_dataArray;

  // Allocate CUDA array in device memory
  //hipChannelFormatDesc channelDesc = hipCreateChannelDesc(
  //    32, 0, 0, 0, hipChannelFormatKindUnsigned);

  int e = (int)sizeof(unsigned) * 8;
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc(e, 0, 0, 0, hipChannelFormatKindUnsigned);

  hipExtent extent = make_hipExtent(nx, ny, nz);
  hipMalloc3DArray(
      &d_dataArray,
      &channelDesc,
      extent,
      hipArrayDefault
  );

  // Copy host camparams to device memory
  hipMemcpy3DParms parms = {0};
  parms.srcPtr = make_hipPitchedPtr(
      h_data,
      nx * sizeof(unsigned), nx, ny
  );
  parms.dstArray = d_dataArray;
  parms.extent = extent;
  parms.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&parms);

  // Set texture reference parameters
  data_tex.addressMode[0] = hipAddressModeWrap;
  data_tex.addressMode[1] = hipAddressModeWrap;
  data_tex.addressMode[2] = hipAddressModeWrap;
  data_tex.filterMode = hipFilterModePoint;
  data_tex.normalized = true;

  // Bind the array to the texture reference
  hipBindTextureToArray(data_tex, d_dataArray, channelDesc);
}


__global__ void camcor_kernel(unsigned short *output)
{

    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= const_nx || y >= const_ny || z >= const_nz) {
        return;
    }

    //normalize coordinates
    float u = (x + 0.5) / (float) const_nx;
    float v = (y + 0.5) / (float) const_ny;
    float w = (z + 0.5) / (float) const_nz;
    float wp = (z - 0.5) / (float) const_nz;

    float a = tex3D(camparam_tex, x, y, 0);
    float b = tex3D(camparam_tex, x, y, 1);
    float offset = tex3D(camparam_tex, x, y, 2);

    unsigned voxel = tex3D(data_tex, u, v, w);
    unsigned previousvoxel = tex3D(data_tex, u, v, wp);

    //if (x == 1 and y == 1 and z == 1) {
    //     printf("x: %d, y: %d, z: %d \n ", x, y, z);
    //     printf("u: %f, v: %f, w: %f, wp: %f \n ", u, v, w, wp);
    //     printf("a: %f, b: %f, o: %f \n ", a, b, offset);
    //    printf("voxel: %d, \n ", voxel);
    //     printf("previousvoxel: %d \n ", previousvoxel);
    // }

    unsigned int i = z * const_ny * const_nx + y * const_nx + x;
    output[i] = (unsigned short) voxel - offset - 0.9f * a * (1 - expf(-b * (previousvoxel - offset)));
    output[i] = output[i] > 0 ? output[i] : 0;

}


__host__ void camcor_GPU(int nx, int ny, int nz, GPUBuffer &outBuf){

    dim3 dimBlock(16, 16, 1);
    dim3 dimGrid((nx + dimBlock.x - 1) / dimBlock.x,
                 (ny + dimBlock.y - 1) / dimBlock.y,
                 (nz + dimBlock.z - 1) / dimBlock.z);

    camcor_kernel<<<dimGrid, dimBlock>>>((unsigned short *) outBuf.getPtr());
    //CudaCheckError();
}
